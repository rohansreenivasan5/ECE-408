#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"
#define TILE_W1 16
#define TILE_WIDTH 16
__global__ void conv_forward_kernel(int batch_num, float *output, const float *input, const float *mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    S - stride step length
    */

    const int H_out = (H - K) / S + 1;
    const int W_out = (W - K) / S + 1;
    const int W_grid_dim = ceil(1.0 * W_out / TILE_W1); // blocks in grid requried to cover the length of the output
    // (void)H_out; // silence declared but never referenced warning. remove this line when you start working
    // (void)W_out; // silence declared but never referenced warning. remove this line when you start working

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // out_4d(0,0,0,0) = a

#define out_4d(i3, i2, i1, i0) output[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
#define in_4d(i3, i2, i1, i0) input[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
#define mask_4d(i3, i2, i1, i0) mask[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]
#define in_3d(i2, i1, i0) input[(i2) * (H * W) + (i1) * (W) + i0]
    // Insert your GPU convolution kernel code here
    int m = blockIdx.y;
    int h = (blockIdx.z / W_grid_dim) * TILE_W1 + threadIdx.y;
    int w = (blockIdx.z % W_grid_dim) * TILE_W1 + threadIdx.x;
    int b = blockIdx.x;
    float result = 0.0f;
    if (w >= 0 && w < W_out && h >= 0 && h < H_out)
    {
        for (int c = 0; c < C; ++c)
        {
            for (int p = 0; p < K; ++p)
            {
                for (int q = 0; q < K; ++q)
                {
                    result += in_3d(c, (h * S) + p, (w * S) + q) * mask_4d(m, c, p, q);
                }
            }
        }
        out_4d(batch_num, m, h, w) = result;
    }

#undef out_4d
#undef in_4d
#undef mask_4d
}
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    const int H_out = (H - K) / S + 1;
    const int W_out = (W - K) / S + 1;
    int input_size = B * C * H * W;
    int output_size = B * M * H_out * W_out;
    int mask_size = M * C * K * K;

    hipMalloc((void **)device_output_ptr, output_size * sizeof(float));
    hipMalloc((void **)device_mask_ptr, mask_size * sizeof(float));

    hipMalloc((void **)device_input_ptr, input_size * sizeof(float));
    hipMemcpy(*device_input_ptr, host_input, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*device_mask_ptr, host_mask, mask_size * sizeof(float), hipMemcpyHostToDevice);
}

__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Set the kernel dimensions and call the kernel

    const int H_out = (H - K) / S + 1;
    const int W_out = (W - K) / S + 1;

    int W_grid_dim = ceil(W_out / (float)TILE_W1);
    int H_grid = ceil(H_out / (float)TILE_W1);
    int Y = W_grid_dim * H_grid;
    hipStream_t stream[B];
    int STREAM_SIZE = B;
    int batch_dim = (C * H * W) * sizeof(float);

    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 dimGrid(1, M, Y);

    for (int i = 0; i < STREAM_SIZE; i++)
    {
        hipStreamCreate(&stream[i]);
    }

    float *input[STREAM_SIZE];

    for (int i = 0; i < STREAM_SIZE; i++)
    {
        hipMalloc((void **)&input[i], batch_dim);
        hipMemcpyAsync(input[i], ((void *)device_input) + (i * batch_dim), batch_dim, hipMemcpyDeviceToDevice, stream[i]);
        conv_forward_kernel<<<dimGrid, dimBlock, 0, stream[i]>>>(i, device_output, input[i], device_mask, B, M, C, H, W, K, S);
    }

    // for (int i = 0; i < STREAM_SIZE; i++)
    // {
    //     conv_forward_kernel<<<dimGrid, dimBlock, 0, stream[i]>>>(i, device_output, input[i], device_mask, B, M, C, H, W, K, S);
    // }
}

__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Copy the output back to host

    // Free device memory
    const int Height_out = (H - K) / S + 1;
    const int Width_out = (W - K) / S + 1;
    int output_size = B * M * Height_out * Width_out;

    hipMemcpy(host_output, device_output, output_size * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(device_output);
    hipFree(device_input);
    hipFree(device_mask);
}

__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for (int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout << "Device " << dev << " name: " << deviceProp.name << std::endl;
        std::cout << "Computational capabilities: " << deviceProp.major << "." << deviceProp.minor << std::endl;
        std::cout << "Max Global memory size: " << deviceProp.totalGlobalMem << std::endl;
        std::cout << "Max Constant memory size: " << deviceProp.totalConstMem << std::endl;
        std::cout << "Max Shared memory size per block: " << deviceProp.sharedMemPerBlock << std::endl;
        std::cout << "Max threads per block: " << deviceProp.maxThreadsPerBlock << std::endl;
        std::cout << "Max block dimensions: " << deviceProp.maxThreadsDim[0] << " x, " << deviceProp.maxThreadsDim[1] << " y, " << deviceProp.maxThreadsDim[2] << " z" << std::endl;
        std::cout << "Max grid dimensions: " << deviceProp.maxGridSize[0] << " x, " << deviceProp.maxGridSize[1] << " y, " << deviceProp.maxGridSize[2] << " z" << std::endl;
        std::cout << "Warp Size: " << deviceProp.warpSize << std::endl;
    }
}